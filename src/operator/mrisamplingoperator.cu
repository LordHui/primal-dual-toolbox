#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include "mrisamplingoperator.h"

#include <iu/iumath.h>
#include <iu/iuhelpermath.h>
#include <iu/iudefs.h>

#include "definitions.h"
#include "iurandom.h"

template<typename InputType, typename OutputType>
MriSamplingOperator<InputType, OutputType>::MriSamplingOperator() :
    OperatorBase<InputType, OutputType>(1, "MriSamplingOperator")
{
}

template<typename InputType, typename OutputType>
MriSamplingOperator<InputType, OutputType>::~MriSamplingOperator()
{
}

template<typename InputType, typename OutputType>
void MriSamplingOperator<InputType, OutputType>::sizeCheck(
    const InputType & src, const OutputType & dst)
{
  IU_SIZE_CHECK(dst.size(), (this->template getConstant<real_type, 2>(0))->size());
  IU_SIZE_CHECK(src.size(), (this->template getConstant<real_type, 2>(0))->size());
}

template<typename InputType, typename OutputType>
iu::Size<InputType::ndim> MriSamplingOperator<InputType, OutputType>::getInputSize(const OutputType & output)
{
  return output.size();
}

template<typename InputType, typename OutputType>
iu::Size<OutputType::ndim> MriSamplingOperator<InputType, OutputType>::getOutputSize(const InputType & input)
{
  return input.size();
}

template<typename InputType, typename OutputType>
void MriSamplingOperator<InputType, OutputType>::executeForward(const InputType & src,
                                                          OutputType & dst)
{
  // extract constants
  auto mask = this->template getConstant<real_type, 2>(0);

  // centered fft
  iu::math::fft::fft2c(src, dst, true);

  // apply sampling mask
  iu::math::complex::multiply(dst, *mask, dst);
}

template<typename InputType, typename OutputType>
void MriSamplingOperator<InputType, OutputType>::executeAdjoint(
    const OutputType & src, InputType & dst)
{
  // extract constants
  auto mask = this->template getConstant<real_type, 2>(0);

  // allocate temporary memory
  InputType tmp(src.size());
  iu::copy(&src, &tmp);

  // apply sampling mask
  iu::math::complex::multiply(tmp, *mask, tmp);

  // centered ifft
  iu::math::fft::ifft2c(tmp, dst, true);
}

template<typename InputType, typename OutputType>
void MriSamplingOperator<InputType, OutputType>::adjointnessCheck()
{
  std::cout << "Test adjointness of operator: " << *this << std::endl;

  InputType u(this->template getConstant<real_type, 2>(0)->size());
  OutputType p(u.size());
  iu::random::fillRandomFloatingNumbers(u);
  iu::random::fillRandomFloatingNumbers(p);
  OutputType Au(p.size());
  InputType Atp(u.size());

  executeForward(u, Au);
  executeAdjoint(p, Atp);

  TEST_ADJOINTNESS(u, Au, p, Atp);
}

// explicit template instantiations
template class MriSamplingOperator<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 2>> ;
template class MriSamplingOperator<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 2>> ;
