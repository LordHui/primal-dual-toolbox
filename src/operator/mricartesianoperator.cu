#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include "mricartesianoperator.h"

#include <iu/iumath.h>
#include <iu/iuhelpermath.h>
#include <iu/iudefs.h>

#include "definitions.h"
#include "iurandom.h"

/** Preparations before FFT: Apply coil sensitivities and perform ifftshift. */
template<typename PixelType>
__global__ void prefft_kernel(
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 3>::KernelData coil_sens,
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 2>::KernelData img,
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 3>::KernelData dst)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  const int c = threadIdx.z + blockIdx.z * blockDim.z;

  const int height = coil_sens.size_[1];
  const int width = coil_sens.size_[0];

  if (x < width && y < height && c < coil_sens.size_[2])
  {
    // for ifftshift
    int x_mid = (width + 1.f) / 2.f;
    int y_mid = (height + 1.f) / 2.f;

    // ifftshift to get destination idx
    int x_dst = (x + x_mid) % width;
    int y_dst = (y + y_mid) % height;

    dst(x_dst, y_dst, c) = complex_multiply < PixelType > (img(x, y), coil_sens(x, y, c));
  }
}

/** Multiply a 3D complex array with a 2D real-valued array. */
template<typename PixelType>
__global__ void applyMask_kernel(
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 3>::KernelData Au,
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::real_type, 2>::KernelData mask)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  const int c = threadIdx.z + blockIdx.z * blockDim.z;

  if (x < Au.size_[0] && y < Au.size_[1] && c < Au.size_[2])
  {
    Au(x, y, c) = Au(x, y, c) * mask(x, y);
  }
}

/** Preparations before IFFT: Apply sampling mask and perform ifftshift. */
template<typename PixelType>
__global__ void preifft_kernel(
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 3>::KernelData f,
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::real_type, 2>::KernelData mask,
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 3>::KernelData dst)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  const int c = threadIdx.z + blockIdx.z * blockDim.z;

  const int height = f.size_[1];
  const int width = f.size_[0];

  if (x < width && y < height && c < f.size_[2])
  {
    // for ifftshift
    int x_mid = (width + 1.f) / 2.f;
    int y_mid = (height + 1.f) / 2.f;

    // ifftshift to get destination idx
    int x_dst = (x + x_mid) % width;
    int y_dst = (y + y_mid) % height;

    dst(x_dst, y_dst, c) = f(x, y, c) * mask(x, y);
  }
}

/** Combine image with coil sensitivity maps (complex conjugate) and store it in a destination image. */
template<typename PixelType>
__global__ void combineImg_kernel(
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 3>::KernelData img,
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 3>::KernelData coil_sens,
    struct iu::LinearDeviceMemory<typename iu::type_trait<PixelType>::complex_type, 2>::KernelData dst)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  const int coils = coil_sens.size_[2];
  const int height = coil_sens.size_[1];
  const int width = coil_sens.size_[0];

  if (x < width && y < height)
  {
    typename iu::type_trait<PixelType>::complex_type sum =
        iu::type_trait<PixelType>::make_complex(0);

    for (int c = 0; c < coils; c++)
    {
      sum += complex_multiply_conjugate < PixelType
          > (img(x, y, c), coil_sens(x, y, c));
    }

    dst(x, y) = sum;
  }
}

/** Crop image and store it in a new variable according to given indices in
  x (phase-encoding) and y (frequency-encoding) direction. */
template<typename InputType>
__global__ void cropFOV_kernel(
    struct InputType::KernelData img,
    struct InputType::KernelData cropped,
    unsigned int FE_start_idx, unsigned int PE_start_idx)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < cropped.size_[0] && y < cropped.size_[1])
  {
    cropped(x, y) = img(x + PE_start_idx, y + FE_start_idx);
  }
}

/** Copy image into a padded image according to given indices in
  x (phase-encoding) and y (frequency-encoding) direction. */
template<typename InputType>
__global__ void padFOV_kernel(
    struct InputType::KernelData img,
    struct InputType::KernelData padded,
    unsigned int FE_start_idx, unsigned int PE_start_idx)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < img.size_[0] && y < img.size_[1])
  {
    padded(x + PE_start_idx, y + FE_start_idx) = img(x, y);
  }
}

template<typename InputType, typename OutputType>
MriCartesianOperator<InputType, OutputType>::MriCartesianOperator() :
    OperatorBase<InputType, OutputType>(2, "MriCartesianOperator")
{
}

template<typename InputType, typename OutputType>
MriCartesianOperator<InputType, OutputType>::~MriCartesianOperator()
{
}

template<typename InputType, typename OutputType>
void MriCartesianOperator<InputType, OutputType>::sizeCheck(
    const InputType & src, const OutputType & dst)
{
  IU_SIZE_CHECK(dst.size(), (this->template getConstant<complex_type, 3>(0))->size());
  IU_SIZE_CHECK(src.size(), (this->template getConstant<real_type, 2>(1))->size());
  IU_SIZE_CHECK(src.size(), iu::Size<2>( { dst.size()[0], dst.size()[1] }));
}

template<typename InputType, typename OutputType>
iu::Size<InputType::ndim> MriCartesianOperator<InputType, OutputType>::getInputSize(const OutputType& output)
{
  iu::Size<2> input_size({output.size()[0], output.size()[1]});
  return input_size;
}

template<typename InputType, typename OutputType>
iu::Size<OutputType::ndim> MriCartesianOperator<InputType, OutputType>::getOutputSize(const InputType& input)
{
  iu::Size<OutputType::ndim> size = this->template getConstant<complex_type, 3>(0)->size();
  return size;
}

template<typename InputType, typename OutputType>
void MriCartesianOperator<InputType, OutputType>::executeForward(const InputType & src,
                                                          OutputType & dst)
{
  // extract constants
  auto coil_sens = this->template getConstant<complex_type, 3>(0);
  auto mask = this->template getConstant<real_type, 2>(1);

  // temporary variable
  iu::LinearDeviceMemory<complex_type, 3> kspace(dst.size());

  // output = A(u)
  dim3 dimBlock(COMMON_BLOCK_SIZE_3D_X, COMMON_BLOCK_SIZE_3D_Y,
  COMMON_BLOCK_SIZE_3D_Z);
  dim3 dimGrid(iu::divUp(dst.size()[0], dimBlock.x),
               iu::divUp(dst.size()[1], dimBlock.y),
               iu::divUp(dst.size()[2], dimBlock.z));

  // perform multiplication with coil sensitivity profiles and ifftshift2
  prefft_kernel<pixel_type> <<<dimGrid, dimBlock>>>(*coil_sens, src, dst);
  IU_CUDA_CHECK;

  // perform fft2 and scale with 1/sqrt(elements)
  iu::math::fft::fft2(dst, kspace, true);

  // perform fftshift2
  iu::math::fft::fftshift2(kspace, dst);

  // apply sampling mask
  applyMask_kernel<pixel_type> <<<dimGrid, dimBlock>>>(dst, *mask);
  IU_CUDA_CHECK;
}

template<typename InputType, typename OutputType>
void MriCartesianOperator<InputType, OutputType>::executeAdjoint(
    const OutputType & src, InputType & dst)
{
  // extract constants
  auto coil_sens = this->template getConstant<complex_type, 3>(0);
  auto mask = this->template getConstant<real_type, 2>(1);

  // temporary variables
  iu::LinearDeviceMemory<complex_type, 3> kspace1(src.size());
  iu::LinearDeviceMemory<complex_type, 3> kspace2(src.size());

  // output = A^H(f)
  dim3 dimBlock(COMMON_BLOCK_SIZE_3D_X, COMMON_BLOCK_SIZE_3D_Y,
  COMMON_BLOCK_SIZE_3D_Z);
  dim3 dimGrid(iu::divUp(src.size()[0], dimBlock.x),
               iu::divUp(src.size()[1], dimBlock.y),
               iu::divUp(src.size()[2], dimBlock.z));

  // perform ifftshift and apply sampling mask
  preifft_kernel<pixel_type> <<<dimGrid, dimBlock>>>(src, *mask, kspace1);
  IU_CUDA_CHECK;

  // perform ifft2 and scale with 1/sqrt(elements)
  iu::math::fft::ifft2(kspace1, kspace2, true);

  // perform fftshift2
  iu::math::fft::fftshift2(kspace2, kspace1);

  // multiply kspace result with coil sensitivity profiles and add up the
  // single channels
  dimGrid = dim3(iu::divUp(src.size()[0], dimBlock.x),
                 iu::divUp(src.size()[1], dimBlock.y), 1);
  combineImg_kernel<pixel_type> <<<dimGrid, dimBlock>>>(kspace1, *coil_sens,
                                                            dst);
  IU_CUDA_CHECK;
}

template<typename InputType, typename OutputType>
void MriCartesianOperator<InputType, OutputType>::adjointnessCheck()
{
  InputType u(this->template getConstant<real_type, 2>(1)->size());
  OutputType p(this->template getConstant<complex_type, 3>(0)->size());
  iu::random::fillRandomFloatingNumbers(u);
  iu::random::fillRandomFloatingNumbers(p);
  OutputType Au(p.size());
  InputType Atp(u.size());

  executeForward(u, Au);
  executeAdjoint(p, Atp);

  std::cout << "Test adjointness of operator:" << *this << std::endl;
  TEST_ADJOINTNESS(u, Au, p, Atp);
}

////////////////////////////////////////////////////////////////////////////////
template<typename InputType, typename OutputType>
MriCartesianRemoveROOSOperator<InputType, OutputType>::MriCartesianRemoveROOSOperator() :
    OperatorBase<InputType, OutputType>(2, "MriCartesianRemoveROOSOperator")
{
}

template<typename InputType, typename OutputType>
MriCartesianRemoveROOSOperator<InputType, OutputType>::~MriCartesianRemoveROOSOperator()
{
}

template<typename InputType, typename OutputType>
iu::Size<InputType::ndim> MriCartesianRemoveROOSOperator<InputType, OutputType>::getInputSize(const OutputType& output)
{
  iu::Size<2> input_size({output.size()[0], output.size()[1]/2});
  return input_size;
}

template<typename InputType, typename OutputType>
iu::Size<OutputType::ndim> MriCartesianRemoveROOSOperator<InputType, OutputType>::getOutputSize(const InputType& input)
{
  iu::Size<OutputType::ndim> size = this->template getConstant<complex_type, 3>(0)->size();
  return size;
}

template<typename InputType, typename OutputType>
void MriCartesianRemoveROOSOperator<InputType, OutputType>::sizeCheck(
    const InputType & src, const OutputType & dst)
{
  iu::Size<2> size = src.size();
  size[1] *= 2;
  IU_SIZE_CHECK(dst.size(), (this->template getConstant<complex_type, 3>(0))->size());
  IU_SIZE_CHECK(size, (this->template getConstant<real_type, 2>(1))->size());
}

template<typename InputType, typename OutputType>
void MriCartesianRemoveROOSOperator<InputType, OutputType>::executeForward(const InputType & src,
                                                          OutputType & dst)
{
  // extract constants
  auto coil_sens = this->template getConstant<complex_type, 3>(0);
  auto mask = this->template getConstant<real_type, 2>(1);

  // temporary variable
  iu::LinearDeviceMemory<complex_type, 3> kspace(dst.size());
  iu::Size<2> size = src.size();
  size[1] *= 2;
  InputType src_pad(size);

  // init padded image with zeros
  iu::math::fill(src_pad, iu::type_trait<typename InputType::pixel_type>::make(0));

  // Pad in read-out direction
  dim3 dimBlockSmall(COMMON_BLOCK_SIZE_3D_X, COMMON_BLOCK_SIZE_3D_Y);
  dim3 dimGridSmall(iu::divUp(src.size()[0], dimBlockSmall.x),
                    iu::divUp(src.size()[1], dimBlockSmall.y));
  unsigned int FE_idx = size[1] * 0.25f + 1;
  padFOV_kernel<InputType><<<dimGridSmall, dimBlockSmall>>>(src, src_pad, FE_idx, 0);
  IU_CUDA_CHECK;

  // output = A(u)
  dim3 dimBlock(COMMON_BLOCK_SIZE_3D_X, COMMON_BLOCK_SIZE_3D_Y,
  COMMON_BLOCK_SIZE_3D_Z);
  dim3 dimGrid(iu::divUp(dst.size()[0], dimBlock.x),
               iu::divUp(dst.size()[1], dimBlock.y),
               iu::divUp(dst.size()[2], dimBlock.z));

  // perform multiplication with coil sensitivity profiles and ifftshift2
  prefft_kernel<pixel_type> <<<dimGrid, dimBlock>>>(*coil_sens, src_pad, dst);
  IU_CUDA_CHECK;

  // perform fft2 and scale with 1/sqrt(elements)
  iu::math::fft::fft2(dst, kspace, true);

  // perform fftshift2
  iu::math::fft::fftshift2(kspace, dst);

  // apply sampling mask
  applyMask_kernel<pixel_type> <<<dimGrid, dimBlock>>>(dst, *mask);
  IU_CUDA_CHECK;
}

template<typename InputType, typename OutputType>
void MriCartesianRemoveROOSOperator<InputType, OutputType>::executeAdjoint(
    const OutputType & src, InputType & dst)
{
  // extract constants
  auto coil_sens = this->template getConstant<complex_type, 3>(0);
  auto mask = this->template getConstant<real_type, 2>(1);

  // temporary variables
  iu::LinearDeviceMemory<complex_type, 3> kspace1(src.size());
  iu::LinearDeviceMemory<complex_type, 3> kspace2(src.size());
  iu::Size<2> size = dst.size();
  size[1] *= 2;
  InputType dst_pad(size);
  const unsigned int FE_idx = size[1] * 0.25f + 1;

  // output = A^H(f)
  dim3 dimBlock(COMMON_BLOCK_SIZE_3D_X, COMMON_BLOCK_SIZE_3D_Y,
  COMMON_BLOCK_SIZE_3D_Z);
  dim3 dimGrid(iu::divUp(src.size()[0], dimBlock.x),
               iu::divUp(src.size()[1], dimBlock.y),
               iu::divUp(src.size()[2], dimBlock.z));

  // perform ifftshift and apply sampling mask
  preifft_kernel<pixel_type> <<<dimGrid, dimBlock>>>(src, *mask, kspace1);
  IU_CUDA_CHECK;

  // perform ifft2 and scale with 1/sqrt(elements)
  iu::math::fft::ifft2(kspace1, kspace2, true);

  // perform fftshift2
  iu::math::fft::fftshift2(kspace2, kspace1);

  // multiply kspace result with coil sensitivity profiles and add up the
  // single channels
  dimGrid = dim3(iu::divUp(src.size()[0], dimBlock.x),
                 iu::divUp(src.size()[1], dimBlock.y), 1);
  combineImg_kernel<pixel_type> <<<dimGrid, dimBlock>>>(kspace1, *coil_sens,
                                                            dst_pad);
  IU_CUDA_CHECK;

  // Remove read-out oversampling
  dimBlock = dim3(COMMON_BLOCK_SIZE_3D_X, COMMON_BLOCK_SIZE_3D_Y, 1);
  dimGrid = dim3(iu::divUp(dst.size()[0], dimBlock.x),
                 iu::divUp(dst.size()[1], dimBlock.y), 1);
  cropFOV_kernel<InputType><<<dimGrid, dimBlock>>>(dst_pad, dst, FE_idx, 0);
  IU_CUDA_CHECK;
}

template<typename InputType, typename OutputType>
void MriCartesianRemoveROOSOperator<InputType, OutputType>::adjointnessCheck()
{
  std::cout << "Test adjointness of operator: " << *this << std::endl;

  InputType u(getInputSize(*this->template getConstant<complex_type, 3>(0)));
  OutputType p(this->template getConstant<complex_type, 3>(0)->size());
  iu::random::fillRandomFloatingNumbers(u);
  iu::random::fillRandomFloatingNumbers(p);
  OutputType Au(p.size());
  InputType Atp(u.size());

  executeForward(u, Au);
  executeAdjoint(p, Atp);

  TEST_ADJOINTNESS(u, Au, p, Atp);
}


// explicit template instantiations
template class MriCartesianOperator<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 3>> ;
template class MriCartesianOperator<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 3>> ;

template class MriCartesianRemoveROOSOperator<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 3>> ;
template class MriCartesianRemoveROOSOperator<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 3>> ;

