#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include "mriradialoperator.h"

#include <iu/iumath.h>
#include <iu/iuhelpermath.h>
#include <iu/iudefs.h>

#include "definitions.h"
#include "iurandom.h"

template<typename InputType, typename OutputType>
MriRadialOperator<InputType, OutputType>::MriRadialOperator(const OpConfigDict &config) :
    OperatorBase<InputType, OutputType>(3, "MriRadialOperator", config, 4, 4)
{
  osf_ = this->getConfigDouble("osf");
  sector_width_ = this->getConfigInt("sector_width");
  kernel_width_ = this->getConfigInt("kernel_width");
  img_dim_ = this->getConfigInt("img_dim");
}

template<typename InputType, typename OutputType>
MriRadialOperator<InputType, OutputType>::~MriRadialOperator()
{
}

template<typename InputType, typename OutputType>
void MriRadialOperator<InputType, OutputType>::sizeCheck(
    const InputType & src, const OutputType & dst)
{
  IU_SIZE_CHECK(src.size(), iu::Size<2>({img_dim_, img_dim_}));
  const int nCh = dst.size()[1];
  const int kspace_dim = dst.size()[0];
  IU_SIZE_CHECK((this->template getConstant<real_type, 2>(0))->size(), iu::Size<2>({kspace_dim, 2}));
  IU_SIZE_CHECK((this->template getConstant<real_type, 2>(1))->size(), iu::Size<2>({kspace_dim, 1}));
  IU_SIZE_CHECK((this->template getConstant<complex_type, 3>(2))->size(), iu::Size<3>({img_dim_,  img_dim_, nCh}));
}

template<typename InputType, typename OutputType>
iu::Size<InputType::ndim> MriRadialOperator<InputType, OutputType>::getInputSize(const OutputType& output)
{
  iu::Size<2> input_size({img_dim_, img_dim_});
  return input_size;
}

template<typename InputType, typename OutputType>
iu::Size<OutputType::ndim> MriRadialOperator<InputType, OutputType>::getOutputSize(const InputType& input)
{
  auto iu_trajectory = this->template getConstant<real_type, 2>(0);
  auto iu_sensitivities = this->template getConstant<complex_type, 3>(2);
  const int kspace_dim = iu_trajectory->size()[0];
  const int nCh = iu_sensitivities->size()[2];
  iu::Size<2> output_size({kspace_dim, nCh});
  return output_size;
}

template<typename InputType, typename OutputType>
void MriRadialOperator<InputType, OutputType>::createNufftOperator()
{
  // extract constants
  auto iu_trajectory = this->template getConstant<real_type, 2>(0);
  auto iu_dcf = this->template getConstant<real_type, 2>(1);
  auto iu_coil_sens = this->template getConstant<complex_type, 3>(2);

  gpuNUFFT::Dimensions img_dims;
  img_dims.width = img_dim_;
  img_dims.height = img_dim_;
  img_dims.depth = 0;

  gpuNUFFT::Array<real_type> trajectory;
  trajectory.dim.length = iu_trajectory->size()[0];
  trajectory.data = iu_trajectory->data();

  gpuNUFFT::Array<real_type> dcf;
  dcf.dim.length = iu_dcf->size()[0];
  dcf.data = iu_dcf->data();

  gpuNUFFT::Array<complex_type> coil_sens;
  coil_sens.dim = img_dims;
  coil_sens.dim.channels = iu_coil_sens->size()[2];
  coil_sens.data = iu_coil_sens->data();

  gpuNUFFT::GpuNUFFTOperatorFactory factory(true,true,true);
  nufft_op_.reset(factory.createGpuNUFFTOperator(trajectory, dcf, coil_sens, kernel_width_, sector_width_, osf_, img_dims));
}

template<typename InputType, typename OutputType>
void MriRadialOperator<InputType, OutputType>::executeForward(const InputType & src,
                                                          OutputType & dst)
{
  if (nufft_op_ == nullptr)
  {
    createNufftOperator();
  }

  gpuNUFFT::Dimensions img_dims;
  img_dims.width = img_dim_;
  img_dims.height = img_dim_;
  img_dims.depth = 0;

  gpuNUFFT::Array<complex_type> img;
  img.dim = img_dims;
  img.data = const_cast<complex_type*>(src.data());

  gpuNUFFT::Array<complex_type> kspace;
  kspace.dim.length = dst.size()[0];
  kspace.dim.channels = dst.size()[1];
  kspace.data = dst.data();

  nufft_op_->performForwardGpuNUFFT(img, kspace);
}

template<typename InputType, typename OutputType>
void MriRadialOperator<InputType, OutputType>::executeAdjoint(
    const OutputType & src, InputType & dst)
{
  if (nufft_op_ == nullptr)
  {
    createNufftOperator();
  }

  gpuNUFFT::Dimensions img_dims;
  img_dims.width = img_dim_;
  img_dims.height = img_dim_;
  img_dims.depth = 0;

  gpuNUFFT::Array<complex_type> img;
  img.dim = img_dims;
  img.data = dst.data();

  gpuNUFFT::Array<complex_type> kspace;
  kspace.dim.length = src.size()[0];
  kspace.dim.channels = src.size()[1];
  kspace.data = const_cast<complex_type*>(src.data());

  nufft_op_->performGpuNUFFTAdj(kspace, img);
}

template<typename InputType, typename OutputType>
void MriRadialOperator<InputType, OutputType>::adjointnessCheck()
{
  iu::Size<2> u_size({img_dim_, img_dim_});
  iu::Size<2> p_size({this->template getConstant<real_type, 2>(1)->size()[0], this->template getConstant<complex_type, 3>(2)->size()[2]});
  InputType u(u_size);
  OutputType p(p_size);
  iu::random::fillRandomFloatingNumbers(u);
  iu::random::fillRandomFloatingNumbers(p);
  OutputType Au(p.size());
  InputType Atp(u.size());

  executeForward(u, Au);
  executeAdjoint(p, Atp);

  std::cout << "Test adjointness of operator:" << *this << std::endl;
  TEST_ADJOINTNESS(u, Au, p, Atp);
}

// explicit template instantiations
template class MriRadialOperator<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 2>> ;
//template class MriRadialOperator<iu::LinearDeviceMemory<double2, 2>,
//    iu::LinearDeviceMemory<double2, 2>> ;
