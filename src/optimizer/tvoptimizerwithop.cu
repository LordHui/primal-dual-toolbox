#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include "tvoptimizerwithop.h"

#include <iu/iucutil.h>
#include <iu/iumath.h>
#include <iu/iuhelpermath.h>

#include "definitions.h"
#include "tgv_kernels.cuh"

template<typename InputType, typename OutputType>
TvOptimizerWithOp<InputType, OutputType>::TvOptimizerWithOp() :
    op_(NULL)
{
}

template<typename InputType, typename OutputType>
TvOptimizerWithOp<InputType, OutputType>::~TvOptimizerWithOp()
{
}

template<typename InputType, typename OutputType>
InputType* TvOptimizerWithOp<InputType, OutputType>::getResult()
{
	if (u_ == nullptr)
	{
		// throw exception
		std::stringstream ss;
		ss << "There is no result yet!";
		throw IuException(ss.str(), __FILE__, __FUNCTION__, __LINE__);
	}
	return u_.get();
}

template<typename InputType, typename OutputType>
void TvOptimizerWithOp<InputType, OutputType>::setOperator(
    const std::shared_ptr<OperatorBase<InputType, OutputType> >& op)
{
  op_ = op;
}

template<typename InputType, typename OutputType>
void TvOptimizerWithOp<InputType, OutputType>::setInput0(const InputType &input)
{
  // primal variables
  u_.reset(new InputType(input.size()));
  u__.reset(new InputType(input.size()));

  // dual variables
  p_.init(input.size());

  // copy input to u
  iu::copy(&input, u_.get());
  iu::copy(&input, u__.get());

}

template<typename InputType, typename OutputType>
void TvOptimizerWithOp<InputType, OutputType>::setNoisyData(
    const std::shared_ptr<OutputType> &f)
{
  // set data
  f_ = f;

  // init dual variable r
  r_.reset(new OutputType(f->size()));
  iu::math::fill(*r_, iu::type_trait<typename OutputType::pixel_type>::make(0));
}

template<typename InputType, typename OutputType>
void TvOptimizerWithOp<InputType, OutputType>::solve(bool verbose)
{
  if (op_ == nullptr)
  {
    // throw exception
    std::stringstream ss;
    ss << "No operator is set!";
    throw IuException(ss.str(), __FILE__, __FUNCTION__, __LINE__);
  }

  if (f_ == nullptr)
  {
    // throw exception
    std::stringstream ss;
    ss << "No noisy data is set!";
    throw IuException(ss.str(), __FILE__, __FUNCTION__, __LINE__);
  }

  if (u_ == nullptr)
  {
    std::cout << "Initialization with A^T(f)" << std::endl;
    InputType input0(op_->getInputSize(*f_));
    op_->adjoint(*f_, input0);
    this->setInput0(input0);
  }

  dim3 dimBlock(COMMON_BLOCK_SIZE_2D_X, COMMON_BLOCK_SIZE_2D_Y);
  dim3 dimGrid(iu::divUp(u_->size()[0], dimBlock.x),
               iu::divUp(u_->size()[1], dimBlock.y));

  dim3 dimBlock1d(COMMON_BLOCK_SIZE_2D_X * COMMON_BLOCK_SIZE_2D_Y);
  dim3 dimGrid1d(iu::divUp(r_->size().numel(), dimBlock.x));

  real_type tau = 1.0 / 8.0;
  real_type sigma = 1.0 / 8.0;
  real_type theta = 1.0;

  iu::IuCudaTimer timer;
  timer.start();

  std::cout << params_ << std::endl;

  for (int k = 0; k <= params_.max_iter; k++)
  {
    if (!(k % params_.check))
    {
      std::cout << "iter=" << k << " lambda=" << params_.lambda << std::endl;
    }

    // pre-compute K(u_)
    OutputType tmp(f_->size());
    op_->forward(*u__, tmp);

    // Compute K(u_) - f_ and store it in tmp
    iu::math::addWeighted(tmp, iu::type_trait < output_pixel_type > ::make(1.0),
                          *f_,
                          iu::type_trait < output_pixel_type > ::make(-1.0),
                          tmp);

    // dual step and projection on r
    TGV_prox_r_kernel<OutputType> <<<dimGrid1d, dimBlock1d>>>(*r_, tmp, sigma, params_.lambda);
    IU_CUDA_CHECK;

    // dual step and projection on p
    TV_dual_p_kernel<InputType> <<<dimGrid, dimBlock>>>(p_, *u__, sigma);
    IU_CUDA_CHECK;

    // pre-compute K*(r) and store it in u__
    op_->adjoint(*r_, *u__);

    // primal step on u
    TGV_primal_u_noprox_kernel<InputType> <<<dimGrid, dimBlock>>>(*u_, *u__, p_,
                                                                  tau, theta);
    IU_CUDA_CHECK;
  }

  std::cout << "reconstruction time " << timer.elapsed() << std::endl;
}

// explicit template instantiations
template class TvOptimizerWithOp<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 3>> ;
template class TvOptimizerWithOp<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 2>> ;
template class TvOptimizerWithOp<iu::LinearDeviceMemory<float, 2>,
    iu::LinearDeviceMemory<float, 3>> ;
template class TvOptimizerWithOp<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 3>> ;
template class TvOptimizerWithOp<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 2>> ;
template class TvOptimizerWithOp<iu::LinearDeviceMemory<float, 2>,
    iu::LinearDeviceMemory<float, 2>> ;
template class TvOptimizerWithOp<iu::LinearDeviceMemory<double, 2>,
    iu::LinearDeviceMemory<double, 2>> ;
template class TvOptimizerWithOp<iu::LinearDeviceMemory<double, 2>,
    iu::LinearDeviceMemory<double, 3>> ;
