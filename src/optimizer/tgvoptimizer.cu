#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include "tgvoptimizer.h"

#include <iu/iucutil.h>
#include <iu/iuhelpermath.h>

#include "definitions.h"
#include "tgv_kernels.cuh"

template<typename PixelType>
TgvParameters<PixelType>::TgvParameters() :
    alpha0(1), alpha1(1), lambda(1), max_iter(1), check(100)
{
}

template<typename PixelType>
TgvParameters<PixelType>::~TgvParameters()
{
}

template<typename InputType, typename OutputType>
TgvOptimizer<InputType, OutputType>::TgvOptimizer()
{
}

template<typename InputType, typename OutputType>
TgvOptimizer<InputType, OutputType>::~TgvOptimizer()
{
}

template<typename InputType, typename OutputType>
InputType* TgvOptimizer<InputType, OutputType>::getResult()
{
  return u_.get();
}

template<typename InputType, typename OutputType>
void TgvOptimizer<InputType, OutputType>::setInput0(const InputType &input)
{
  // primal variables
  u_.reset(new InputType(input.size()));
  u__.reset(new InputType(input.size()));
  v_.init(input.size());
  v__.init(input.size());

  // dual variables
  p_.init(input.size());
  q_.init(input.size());

  // copy input to u
  iu::copy(&input, u_.get());
  iu::copy(&input, u__.get());

}

template<typename InputType, typename OutputType>
void TgvOptimizer<InputType, OutputType>::setNoisyData(
    const std::shared_ptr<OutputType> &f)
{
  // set data
  f_ = f;
}

template<typename InputType, typename OutputType>
void TgvOptimizer<InputType, OutputType>::solve(bool verbose)
{
  if (f_ == nullptr)
  {
    // throw exception
    std::stringstream ss;
    ss << "No noisy data is set!";
    throw IuException(ss.str(), __FILE__, __FUNCTION__, __LINE__);
  }

  if (u_ == nullptr)
  {
    std::cout << "Initialization with f" << std::endl;
    iu::Size<2> size(f_->size()[0], f_->size()[1]);
    this->setInput0(*f_);
  }

  dim3 dimBlock(COMMON_BLOCK_SIZE_2D_X, COMMON_BLOCK_SIZE_2D_Y);
  dim3 dimGrid(iu::divUp(u_->size()[0], dimBlock.x),
               iu::divUp(u_->size()[1], dimBlock.y));

  real_type tau = 1.0 / sqrt(12.0);
  real_type sigma = 1.0 / sqrt(12.0);
  real_type theta = 1.0;

  iu::IuCudaTimer timer;
  timer.start();

  std::cout << params_ << std::endl;

  for (unsigned int k = 0; k <= params_.max_iter; k++)
  {
    if (!(k % params_.check))
    {
      std::cout << "iter=" << k << std::endl;
    }

    // primal step on u
    TV_primal_u_kernel<InputType> <<<dimGrid, dimBlock>>>(*u_, *u__, *f_, p_, tau,
                                                          params_.lambda, theta);

    IU_CUDA_CHECK;

    // primal step on v
    TGV_primal_v_kernel<InputType> <<<dimGrid, dimBlock>>>(v_, v__, p_, q_, tau,
                                                           theta);
    IU_CUDA_CHECK;

    // dual step and projection on p
    TGV_dual_p_kernel<InputType> <<<dimGrid, dimBlock>>>(p_, *u__, v__, sigma,
                                                         params_.alpha1);
    IU_CUDA_CHECK;

    // dual step and prox on q
    TGV_dual_q_kernel<InputType> <<<dimGrid, dimBlock>>>(q_, v__, sigma,
                                                         params_.alpha0);

  }

  std::cout << "reconstruction time " << timer.elapsed() << std::endl;
}

// explicit template instantiations
template class TgvOptimizer<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 2>> ;
template class TgvOptimizer<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 2>> ;
template class TgvOptimizer<iu::LinearDeviceMemory<float, 2>,
    iu::LinearDeviceMemory<float, 2>> ;
template class TgvOptimizer<iu::LinearDeviceMemory<double, 2>,
    iu::LinearDeviceMemory<double, 2>> ;

template class TgvParameters<float>;
template class TgvParameters<double>;
