#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include "tvoptimizer.h"

#include <iu/iucutil.h>

#include "definitions.h"
#include "tgv_kernels.cuh"

template<typename PixelType>
TvParameters<PixelType>::TvParameters() :
    lambda(1), max_iter(1), check(100)
{
}

template<typename PixelType>
TvParameters<PixelType>::~TvParameters()
{
}

template<typename InputType, typename OutputType>
TvOptimizer<InputType, OutputType>::TvOptimizer()
{
}

template<typename InputType, typename OutputType>
TvOptimizer<InputType, OutputType>::~TvOptimizer()
{
}

template<typename InputType, typename OutputType>
InputType* TvOptimizer<InputType, OutputType>::getResult()
{
  return u_.get();
}

template<typename InputType, typename OutputType>
void TvOptimizer<InputType, OutputType>::setInput0(const InputType &input)
{
  // primal variables
  u_.reset(new InputType(input.size()));
  u__.reset(new InputType(input.size()));

  // dual variables
  p_.init(input.size());

  // copy input to u
  iu::copy(&input, u_.get());
  iu::copy(&input, u__.get());

}

template<typename InputType, typename OutputType>
void TvOptimizer<InputType, OutputType>::setNoisyData(
    const std::shared_ptr<OutputType> &f)
{
  // set data
  f_ = f;
}

template<typename InputType, typename OutputType>
void TvOptimizer<InputType, OutputType>::solve(bool verbose)
{
  if (f_ == nullptr)
  {
    // throw exception
    std::stringstream ss;
    ss << "No noisy data is set!";
    throw IuException(ss.str(), __FILE__, __FUNCTION__, __LINE__);
  }

  if (u_ == nullptr)
  {
    std::cout << "Initialization with f" << std::endl;
    iu::Size<2> size(f_->size()[0], f_->size()[1]);
    this->setInput0(*f_);
  }

  dim3 dimBlock(COMMON_BLOCK_SIZE_2D_X, COMMON_BLOCK_SIZE_2D_Y);
  dim3 dimGrid(iu::divUp(u_->size()[0], dimBlock.x),
               iu::divUp(u_->size()[1], dimBlock.y));

  real_type tau = 1.0 / sqrt(8.0);
  real_type sigma = 1.0 / sqrt(8.0);
  real_type theta = 1.0;

  iu::IuCudaTimer timer;
  timer.start();

  std::cout << params_ << std::endl;

  for (unsigned int k = 0; k <= params_.max_iter; k++)
  {
    if (!(k % params_.check))
    {
      std::cout << "iter=" << k << std::endl;
    }

    // dual step and projection on p
    TV_dual_p_kernel<InputType> <<<dimGrid, dimBlock>>>(p_, *u__, sigma);
    IU_CUDA_CHECK;

    // primal step on u
    TV_primal_u_kernel<InputType> <<<dimGrid, dimBlock>>>(*u_, *u__, *f_, p_, tau,
                                                          params_.lambda, theta);
    IU_CUDA_CHECK;
  }

  std::cout << "reconstruction time " << timer.elapsed() << std::endl;
}

// explicit template instantiations
template class TvOptimizer<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 2>> ;
template class TvOptimizer<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 2>> ;
template class TvOptimizer<iu::LinearDeviceMemory<float, 2>,
    iu::LinearDeviceMemory<float, 2>> ;
template class TvOptimizer<iu::LinearDeviceMemory<double, 2>,
    iu::LinearDeviceMemory<double, 2>> ;

template class TvParameters<float>;
template class TvParameters<double>;
