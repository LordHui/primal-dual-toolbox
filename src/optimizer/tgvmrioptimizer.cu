#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include "tgvmrioptimizer.h"

#include <iu/iucutil.h>
#include <iu/iuhelpermath.h>
#include <iu/iumath.h>

#include "definitions.h"
#include "tgv_kernels.cuh"

template<typename PixelType>
TgvMriParameters<PixelType>::TgvMriParameters() :
    alpha0(1), alpha1(1), max_iter(1), reduction(1), check(100)
{
}

template<typename PixelType>
TgvMriParameters<PixelType>::~TgvMriParameters()
{
}

template<typename InputType, typename OutputType>
TgvMriOptimizer<InputType, OutputType>::TgvMriOptimizer() :
    op_(NULL)
{
}

template<typename InputType, typename OutputType>
TgvMriOptimizer<InputType, OutputType>::~TgvMriOptimizer()
{
}

template<typename InputType, typename OutputType>
InputType* TgvMriOptimizer<InputType, OutputType>::getResult()
{
  return u_.get();
}

template<typename InputType, typename OutputType>
void TgvMriOptimizer<InputType, OutputType>::setOperator(
    const std::shared_ptr<OperatorBase<InputType, OutputType> >& op)
{
  op_ = op;
}

template<typename InputType, typename OutputType>
void TgvMriOptimizer<InputType, OutputType>::setInput0(const InputType &input)
{
  // primal variables
  u_.reset(new InputType(input.size()));
  u__.reset(new InputType(input.size()));
  v_.init(input.size());
  v__.init(input.size());

  // dual variables
  p_.init(input.size());
  q_.init(input.size());

  // copy input to u
  iu::copy(&input, u_.get());
  iu::copy(&input, u__.get());

}

template<typename InputType, typename OutputType>
void TgvMriOptimizer<InputType, OutputType>::setNoisyData(
    const std::shared_ptr<OutputType> &f)
{
  // set data
  f_ = f;

  // init dual variable r
  r_.reset(new OutputType(f->size()));
  iu::math::fill(*r_, iu::type_trait<typename OutputType::pixel_type>::make(0));
}

template<typename InputType, typename OutputType>
void TgvMriOptimizer<InputType, OutputType>::solve(bool verbose)
{
  if (op_ == nullptr)
  {
    // throw exception
    std::stringstream ss;
    ss << "No operator is set!";
    throw IuException(ss.str(), __FILE__, __FUNCTION__, __LINE__);
  }

  if (f_ == nullptr)
  {
    // throw exception
    std::stringstream ss;
    ss << "No noisy data is set!";
    throw IuException(ss.str(), __FILE__, __FUNCTION__, __LINE__);
  }

  if (u_ == nullptr)
  {
    std::cout << "Initialization with A^T(f)" << std::endl;
    InputType input0(op_->getInputSize(*f_));
    op_->adjoint(*f_, input0);
    this->setInput0(input0);
  }

  dim3 dimBlock(COMMON_BLOCK_SIZE_2D_X, COMMON_BLOCK_SIZE_2D_Y);
  dim3 dimGrid(iu::divUp(u_->size()[0], dimBlock.x),
               iu::divUp(u_->size()[1], dimBlock.y));

  dim3 dimBlock1d(COMMON_BLOCK_SIZE_2D_X * COMMON_BLOCK_SIZE_2D_Y);
  dim3 dimGrid1d(iu::divUp(r_->size().numel(), dimBlock.x));

  real_type alpha00 = params_.alpha0;
  real_type alpha10 = params_.alpha1;
  real_type alpha01 = params_.alpha0 * params_.reduction;
  real_type alpha11 = params_.alpha1 * params_.reduction;

  real_type tau = 1.0 / 16.0;
  real_type sigma = 1.0 / 8.0;
  real_type theta = 1.0;

  iu::IuCudaTimer timer;
  timer.start();

  std::cout << params_ << std::endl;

  // convert unsigned int to floating type
  real_type dmax_iter = static_cast<real_type>(params_.max_iter);

  for (unsigned int k = 0; k <= params_.max_iter; k++)
  {
    // convert int to floating type
    real_type dk = static_cast<real_type>(k);

    // update alpha0, alpha1
    real_type alpha0 = exp(
        dk / dmax_iter * log(alpha01)
            + (dmax_iter - dk) / dmax_iter * log(alpha00));
    real_type alpha1 = exp(
        dk / dmax_iter * log(alpha11)
            + (dmax_iter - dk) / dmax_iter * log(alpha10));

    if (!(k % params_.check))
    {
      std::cout << "iter=" << k << " alpha0=" <<  alpha0 << " alpha1=" << alpha1 << std::endl;
    }

    // pre-compute K(u_)
    OutputType tmp(f_->size());
    op_->forward(*u__, tmp);

    // Compute K(u_) - f_ and store it in tmp
    iu::math::addWeighted(tmp, iu::type_trait < output_pixel_type > ::make(1.0),
                          *f_,
                          iu::type_trait < output_pixel_type > ::make(-1.0),
                          tmp);
	  IU_CUDA_CHECK;

    // dual step and projection on r
    TGV_prox_r_kernel<OutputType> <<<dimGrid1d, dimBlock1d>>>(*r_, tmp, sigma);
    IU_CUDA_CHECK;

    // dual step and projection on p
    TGV_dual_p_kernel<InputType> <<<dimGrid, dimBlock>>>(p_, *u__, v__, sigma,
                                                         alpha1);
    IU_CUDA_CHECK;

    // dual step and prox on q
    TGV_dual_q_kernel<InputType> <<<dimGrid, dimBlock>>>(q_, v__, sigma,
                                                         alpha0);
    IU_CUDA_CHECK;

    // pre-compute K*(r) and store it in u__
    op_->adjoint(*r_, *u__);

    // primal step on u
    TGV_primal_u_noprox_kernel<InputType> <<<dimGrid, dimBlock>>>(*u_, *u__, p_,
                                                                  tau, theta);
    IU_CUDA_CHECK;

    // primal step on v
    TGV_primal_v_kernel<InputType> <<<dimGrid, dimBlock>>>(v_, v__, p_, q_, tau,
                                                           theta);
    IU_CUDA_CHECK;

  }

  std::cout << "reconstruction time " << timer.elapsed() << std::endl;
}

// explicit template instantiations
template class TgvMriOptimizer<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 3>> ;
template class TgvMriOptimizer<iu::LinearDeviceMemory<float2, 2>,
    iu::LinearDeviceMemory<float2, 2>> ;
template class TgvMriOptimizer<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 3>> ;
template class TgvMriOptimizer<iu::LinearDeviceMemory<double2, 2>,
    iu::LinearDeviceMemory<double2, 2>> ;
template class TgvMriOptimizer<iu::LinearDeviceMemory<float, 2>,
    iu::LinearDeviceMemory<float, 2>> ;
template class TgvMriOptimizer<iu::LinearDeviceMemory<double, 2>,
    iu::LinearDeviceMemory<double, 2>> ;

template class TgvMriParameters<float>;
template class TgvMriParameters<double>;
