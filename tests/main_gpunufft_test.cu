#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include <iostream>

#include "iurandom.h"
#include <gpuNUFFT_operator_factory.hpp>

int main(int argc, char *argv[])
{
  std::cout << "*****************************" << std::endl;
  std::cout << "Test gpuNUFFT"<< std::endl;
  std::cout << "*****************************" << std::endl;

  const unsigned int nFE = 512;
  const unsigned int nSpokes = 64;
  const unsigned int osf = 2;
  const unsigned int sector_width = 3;
  const unsigned int kernel_width = 3;
  const unsigned int nCh = 32;

  iu::LinearDeviceMemory<float, 2> iu_kSpaceTraj({nFE * nSpokes, 2});
  iu::random::fillRandomFloatingNumbers(iu_kSpaceTraj);

  iu::LinearDeviceMemory<float, 1> iu_dcf({nFE * nSpokes});
  iu::random::fillRandomFloatingNumbers(iu_dcf);

  iu::LinearDeviceMemory<float2, 3> iu_sens({nFE/osf, nFE/osf, nCh});
  iu::random::fillRandomFloatingNumbers(iu_sens);

  iu::LinearDeviceMemory<float2, 2> iu_img({nFE/osf, nFE/osf});
  iu::random::fillRandomFloatingNumbers(iu_img);

  iu::LinearDeviceMemory<float2, 3> iu_kspace({nFE, nSpokes, nCh});
  iu::random::fillRandomFloatingNumbers(iu_kspace);

  gpuNUFFT::Array<float> kSpaceTraj;
  kSpaceTraj.dim.length = nFE * nSpokes;
  kSpaceTraj.data = iu_kSpaceTraj.data();

  gpuNUFFT::Array<float> dcf;
  dcf.dim.length = nFE * nSpokes;
  dcf.data = iu_dcf.data();

  gpuNUFFT::Dimensions img_dims;
  img_dims.width = nFE/osf;
  img_dims.height = nFE/osf;
  img_dims.depth = 0;

  gpuNUFFT::Array<float2> sens;
  sens.dim = img_dims;
  sens.dim.channels = nCh;
  sens.data = iu_sens.data();

  gpuNUFFT::Array<float2> img;
  img.dim = img_dims;
  img.data = iu_img.data();

  gpuNUFFT::Array<float2> kspace;
  kspace.dim.length = nFE * nSpokes;
  kspace.dim.channels = nCh;
  kspace.data = iu_kspace.data();

  gpuNUFFT::GpuNUFFTOperatorFactory factory(false,true,true);
  gpuNUFFT::GpuNUFFTOperator * nufft_op = factory.createGpuNUFFTOperator(
      kSpaceTraj, dcf, sens, kernel_width, sector_width, osf, img_dims);

  nufft_op->performForwardGpuNUFFT(img, kspace);
  nufft_op->setSens(sens);
  nufft_op->performForwardGpuNUFFT(img, kspace);

  delete nufft_op;
  return 0;
}
