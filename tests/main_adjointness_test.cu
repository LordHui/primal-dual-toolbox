#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include <iostream>
#include "operator/mricartesianoperator.h"
#include "operator/mrisamplingoperator.h"

#ifdef WITH_GPUNUFFT
  #include "operator/mriradialoperator.h"
#endif

#include "iurandom.h"

void testMriCartesianOperator()
{
  // Create constants needed for operator
  iu::LinearHostMemory<double2, 3> h_coilsens({10,10,5});
  iu::LinearHostMemory<double, 2> h_mask({10,10});
  iu::random::fillRandomFloatingNumbers(h_coilsens);
  iu::random::fillRandomIntNumbers(h_mask, 0, 1);

  // Create operator, add constants and check for adjointness
  MriCartesianOperator<iu::LinearDeviceMemory<double2, 2>, iu::LinearDeviceMemory<double2, 3>> op;
  op.addConstant(h_coilsens);
  op.addConstant(h_mask);
  op.adjointnessCheck();
}

void testMriCartesianRemoveROOSOperator()
{
  // Create constants needed for operator
  iu::LinearHostMemory<double2, 3> h_coilsens({10,10,5});
  iu::LinearHostMemory<double, 2> h_mask({10,10});
  iu::random::fillRandomFloatingNumbers(h_coilsens);
  iu::random::fillRandomIntNumbers(h_mask, 0, 1);

  // Create operator, add constants and check for adjointness
  MriCartesianRemoveROOSOperator<iu::LinearDeviceMemory<double2, 2>, iu::LinearDeviceMemory<double2, 3>> op;
  op.addConstant(h_coilsens);
  op.addConstant(h_mask);
  op.adjointnessCheck();
}

void testMriSamplingOperator()
{
  // Create operator & check adjointness
  iu::LinearHostMemory<double, 2> h_mask({10,10});
  iu::random::fillRandomIntNumbers(h_mask, 0, 1);

  MriSamplingOperator<iu::LinearDeviceMemory<double2, 2>, iu::LinearDeviceMemory<double2, 2>> op;
  op.addConstant(h_mask);
  op.adjointnessCheck();
}

void testMriRadialOperator()
{
#ifdef WITH_GPUNUFFT
  // Setup config parameters
  OpConfigDict config;
  config["img_dim"] = "256";
  config["osf"] = "2";
  config["kernel_width"] = "3";
  config["sector_width"] = "5";

  // Constants
  iu::LinearHostMemory<float2, 3> h_coilsens({256,256, 5});
  iu::LinearHostMemory<float, 2> h_trajectory({256*64, 2});
  iu::LinearHostMemory<float, 2> h_dcf({256*64, 1});

  iu::random::fillRandomFloatingNumbers(h_trajectory);
  iu::random::fillRandomFloatingNumbers(h_dcf);
  iu::random::fillRandomFloatingNumbers(h_coilsens);

  // Create operator & check adjointness
  MriRadialOperator<iu::LinearDeviceMemory<float2, 2>, iu::LinearDeviceMemory<float2, 2>> op(config);
  op.addConstant(h_trajectory);
  op.addConstant(h_dcf);
  op.addConstant(h_coilsens);
  op.adjointnessCheck();
#endif
}

int main(int argc, char *argv[])
{
  std::cout << "*****************************" << std::endl;
  std::cout << "Test adjointness of operators"<< std::endl;
  std::cout << "*****************************" << std::endl;

  testMriCartesianOperator();
  testMriCartesianRemoveROOSOperator();
  testMriSamplingOperator();
  testMriRadialOperator();
  return 0;
}
