#include "hip/hip_runtime.h"
// This file is part of primal-dual-toolbox.
//
// Copyright (C) 2018 Kerstin Hammernik <hammernik at icg dot tugraz dot at>
// Institute of Computer Graphics and Vision, Graz University of Technology
// https://www.tugraz.at/institute/icg/research/team-pock/
//
// primal-dual-toolbox is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// primal-dual-toolbox is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program. If not, see <http://www.gnu.org/licenses/>.

#include <iostream>
#include "iurandom.h"

int main(int argc, char *argv[])
{
  iu::LinearHostMemory<double2, 2> array( { 10, 1 });

  iu::random::fillRandomFloatingNumbers(array);

  for (int i = 0; i < 10; i++)
  {
    std::cout << array.data()[i] << std::endl;
  }

  return 0;
}
